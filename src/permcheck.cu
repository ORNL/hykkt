#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include "permcheck.hpp"
/*
@brief: maps the values in old_val to new_val based on perm

@inputs: Size n of the matrix, perm - desired permutation,
and old_val - the array to be permuted

@outputs: new_val contains the permuted old_val
 */

void fun_map_idx(int n, int* perm, double* old_val, double* new_val)
{
  int numBlocks, blockSize=512;
  numBlocks = (n + blockSize - 1) / blockSize;
  map_idx<<<numBlocks, blockSize>>>(n, perm, old_val, new_val);
}
__global__ void map_idx(int n, int* perm, double* old_val, double* new_val)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < n)
  {
    new_val[i] = old_val[perm[i]];
  }
}
/*
@brief: Selection sorts arr1 and arr2 w/indices
based on increasing value in arr1

@inputs: Size n of the matrix, arr1 - the array that determines the sorting order,
arr2- sorted based on arr1

@outputs: arr1 and arr2 are sorted based on increasing values in arr1
 */

void selection_sort2(int len, int* arr1, int* arr2)
{
  int min_ind;
  int temp;
  for(int i = 0; i < len - 1; i++)
  {
    min_ind = i;
    for(int j = i + 1; j < len; j++)
    {
      if(arr1[j] < arr1[min_ind])
      {
        min_ind = j;
      }
    }
    if(i != min_ind)
    {
      temp          = arr1[i];
      arr1[i]       = arr1[min_ind];
      arr1[min_ind] = temp;
      temp          = arr2[i];
      arr2[i]       = arr2[min_ind];
      arr2[min_ind] = temp;
    }
  }
}
/*
@brief: Permutes the columns in a matrix represented by rows and cols

@inputs: Size n of the matrix, rows and cols - representing the matrix,
rev_perm - the permutation to be applied

@outputs: perm_cols is now the permuted column array and perm_map stores
the corresponding indices to facilitate permuting the values
 */
void make_vec_map_c(int n, int* rows, int* cols, int* rev_perm, int* perm_cols, int* perm_map)
{
  int row_s, rowlen;
  for(int i = 0; i < n; i++)
  {
    row_s  = rows[i];
    rowlen = rows[i + 1] - row_s;
    for(int j = 0; j < rowlen; j++)
    {
      perm_map[row_s + j]  = row_s + j;
      perm_cols[row_s + j] = rev_perm[cols[row_s + j]];
    }
    selection_sort2(rowlen, &perm_cols[row_s], &perm_map[row_s]);
  }
}

/*
@brief: Creates a reverse permutate based on a given permutation

@inputs: Size n of the vector, perm - original permutation

@outputs: rev_perm now contains the reverse permutation
 */
void reverse_perm(int n, int* perm, int* rev_perm)
{
  for(int i = 0; i < n; i++)
  {
    rev_perm[perm[i]] = i;
  }
}

/*
@brief: Permutes the rows in a matrix represented by rows and cols

@inputs: Size n of the matrix, rows and cols - representing the matrix,
perm - the permutation to be applied

@outputs: perm_rows and perm_cols are now the permuted rows and column arrays
and perm_map stores the corresponding indices to facilitate permuting the values
 */
void make_vec_map_r(
  int n, int* rows, int* cols, int* perm, int* perm_rows, int* perm_cols, int* perm_map)
{
  perm_rows[0] = 0;
  int count    = 0, idx, row_s, rowlen;
  for(int i = 0; i < n; i++)
  {
    idx              = perm[i];
    row_s            = rows[idx];
    rowlen           = rows[idx + 1] - row_s;
    perm_rows[i + 1] = perm_rows[i] + rowlen;
    for(int j = 0; j < rowlen; j++)
    {
      perm_map[count + j]  = row_s + j;
      perm_cols[count + j] = cols[row_s + j];
    }
    count += rowlen;
  }
}
/*
@brief: Permutes the rows and columns in a matrix represented by rows and cols

@inputs: Size n of the matrix, rows and cols - representing the matrix,
perm - the permutation to be applied on rows, rev_perm permutation to be applied
on the columns

@outputs: perm_rows and perm_cols are now the permuted rows and column arrays
and perm_map stores the corresponding indices to facilitate permuting the values
 */
void make_vec_map_rc(int n, int* rows, int* cols, int* perm, int* rev_perm, int* perm_rows,
  int* perm_cols, int* perm_map)
{
  perm_rows[0] = 0;
  int count    = 0, idx, row_s, rowlen;
  for(int i = 0; i < n; i++)
  {
    idx              = perm[i];
    row_s            = rows[idx];
    rowlen           = rows[idx + 1] - row_s;
    perm_rows[i + 1] = perm_rows[i] + rowlen;
    for(int j = 0; j < rowlen; j++)
    {
      perm_map[count + j]  = row_s + j;
      perm_cols[count + j] = rev_perm[cols[row_s + j]];
    }
    selection_sort2(rowlen, &perm_cols[count], &perm_map[count]);
    count += rowlen;
  }
}
