#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <sys/time.h>
#include <algorithm>
#include "hipsolver.h"
#include <hipsolver.h>
#include <hipsolver.h>
#include <schur_complement_cg.hpp>

/** Brief: Solves the equation JC H^{-1} JC^T x = b
  via Chronopoulous Gear conjugate gradient 
  Input: JC and JCt in csr format, factorized H,
  initial guss x0, rhs b, max iterations,convergence tolerance, 
  dimensions and nonzeros of JC, matrix description,
  handles for cusparse, cusolver, cublas
  Output: x0 is changed to the solution to JH^{-1}J^Tx=b
  Transpose Jc as we will need it repeatedly
*/
void schur_cg(hipsparseSpMatDescr_t matJC, hipsparseSpMatDescr_t matJCt,
    csrcholInfo_t dH, double* x0, double* b, const int itmax, const double tol,
    int n, int m, int nnz, void* buffer_gpu, hipsparseHandle_t handle, 
    hipsolverSpHandle_t handle_cusolver, hipblasHandle_t handle_cublas)
{
  // Start of block - CG setup
  // create constants for multiplication and allocation - happens once
  double               one      = 1.0;
  double               zero     = 0.0;
  double               minusone = -1.0;
  hipsparseDnVecDescr_t vecx     = NULL;
  hipsparseCreateDnVec(&vecx, n, x0, HIP_R_64F);
  hipsparseDnVecDescr_t vecb = NULL;
  hipsparseCreateDnVec(&vecb, n, b, HIP_R_64F);
  // create vectors necessary
  double* ycp = (double*)calloc(m, sizeof(double));
  double *y, *z, *r, *p, *s, *w;
  for(int i = 0; i < m; i++)
  {
    ycp[i] = 0;
  }
  hipMalloc((void**)&y, m * sizeof(double));
  hipMalloc((void**)&z, m * sizeof(double));
  hipMalloc((void**)&r, n * sizeof(double));
  hipMalloc((void**)&w, n * sizeof(double));
  hipMalloc((void**)&p, n * sizeof(double));
  hipMalloc((void**)&s, n * sizeof(double));
  //  Initializing values - happens every iteration
  hipMemcpy(y, ycp, sizeof(double) * (m), hipMemcpyHostToDevice);
  hipMemcpy(z, y, sizeof(double) * (m), hipMemcpyDeviceToDevice);
  hipMemcpy(r, b, sizeof(double) * (n), hipMemcpyDeviceToDevice);
  hipMemcpy(w, b, sizeof(double) * (n), hipMemcpyDeviceToDevice);
  hipMemcpy(p, r, sizeof(double) * (n), hipMemcpyDeviceToDevice);
  hipMemcpy(s, w, sizeof(double) * (n), hipMemcpyDeviceToDevice);
  double timeIO = 0.0;

  //  Allocation - happens once
  hipsparseDnVecDescr_t vecy = NULL;
  hipsparseCreateDnVec(&vecy, m, y, HIP_R_64F);
  hipsparseDnVecDescr_t vecz = NULL;
  hipsparseCreateDnVec(&vecz, m, z, HIP_R_64F);
  hipsparseDnVecDescr_t vecr = NULL;
  hipsparseCreateDnVec(&vecr, n, r, HIP_R_64F);
  hipsparseDnVecDescr_t vecw = NULL;
  hipsparseCreateDnVec(&vecw, n, w, HIP_R_64F);
  hipsparseDnVecDescr_t vecp = NULL;
  hipsparseCreateDnVec(&vecp, n, p, HIP_R_64F);
  hipsparseDnVecDescr_t vecs = NULL;
  hipsparseCreateDnVec(&vecs, n, s, HIP_R_64F);
  struct timeval t1, t2;
  gettimeofday(&t1, 0);
  /* This is zero anyways
  size_t         bufferSizet = 0, bufferSize = 0;
  hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matJCt, vecx, &zero, vecy,
    HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSizet);
  void* buffert = NULL;
  hipMalloc(&buffert, bufferSizet);
  */
  // Start of block - 0 iteration opertaions for Chronopoulos Gear CG (every iteration)
  fun_SpMV(one, matJCt, vecx, zero, vecy);
  cusolverSpDcsrcholSolve(handle_cusolver, m, y, z, dH, buffer_gpu);
  /* This is zero anyways
  hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minusone, matJC, vecz, &one,
    vecr, HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize);
  void* buffer = NULL;
  hipMalloc(&buffer, bufferSize);
  */
  fun_SpMV(minusone, matJC, vecz, one, vecr);
  double gam_i;
#if 0
	double *h_r =(double*)  calloc(n, sizeof(double));
  hipMemcpy(h_r, r, sizeof(double)*(n), hipMemcpyDeviceToHost);
  for(int i=n-10; i<n; i++){
    printf("h_r[%d] = %f\n", i, h_r[i]);
  }
#endif
  hipblasDdot(handle_cublas, n, r, 1, r, 1, &gam_i);
  // printf("Iteration 0 gamma = %f\n", gam_i);
  // product with w=Ar starts here
  fun_SpMV(one, matJCt, vecr, zero, vecy);
  cusolverSpDcsrcholSolve(handle_cusolver, m, y, z, dH, buffer_gpu);
  fun_SpMV(one, matJC, vecz, zero, vecw);
  double beta = 0, delta, alpha, gam_i1;
  hipblasDdot(handle_cublas, n, w, 1, r, 1, &delta);
  alpha           = gam_i / delta;
  double minalpha = -alpha;
  //  printf("Iteration 0 delta = %f, gamma = %f, alpha = %f\n", delta, gam_i, alpha);
  // Start of block - Main CG iteration
  for(int i = 0; i < itmax; i++)
  {
    hipblasDscal(handle_cublas, n, &beta, p, 1);
    hipblasDaxpy(handle_cublas, n, &one, r, 1, p, 1);

    hipblasDscal(handle_cublas, n, &beta, s, 1);
    hipblasDaxpy(handle_cublas, n, &one, w, 1, s, 1);

    hipblasDaxpy(handle_cublas, n, &alpha, p, 1, x0, 1);
    minalpha = -alpha;
    hipblasDaxpy(handle_cublas, n, &minalpha, s, 1, r, 1);

    hipblasDdot(handle_cublas, n, r, 1, r, 1, &gam_i1);
    if(sqrt(gam_i1) < tol)
    {
      printf("Convergence occured at iteration %d\n", i);
      break;
    }
    // product with w=Ar starts here
    fun_SpMV(one, matJCt, vecr, zero, vecy);
    cusolverSpDcsrcholSolve(handle_cusolver, m, y, z, dH, buffer_gpu);
    fun_SpMV(one, matJC, vecz, zero, vecw);

    hipblasDdot(handle_cublas, n, w, 1, r, 1, &delta);
    beta  = gam_i1 / gam_i;
    gam_i = gam_i1;
    alpha = gam_i / (delta - beta * gam_i / alpha);
  }
  gettimeofday(&t2, 0);
  timeIO = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
  printf("time for CG ev(ms). : %16.16f\n", timeIO);
  printf("Error is %32.32g \n", sqrt(gam_i1));
  free(ycp);
  hipFree(y);
  hipFree(z);
  hipFree(r);
  hipFree(w);
  hipFree(p);
  hipFree(s);
  // hipFree(buffert);
  // hipFree(buffer);
}
