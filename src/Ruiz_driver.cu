#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <ctype.h>
#include <string.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <sys/time.h>
#include <algorithm>
#include "hipsolver.h"
#include <hipsolver.h>
#include <hipsolver.h>
#include <iostream>
#include <memory>
#include <string>
#include <tgmath.h>
#include <math.h>
#include <matrix_vector_ops.hpp>

#define ruiz_its 2
#define tol 1e-8

typedef struct {
  int * coo_rows;
  int * coo_cols;
  double * coo_vals;

  int * csr_ia;

  int n;
  int m;
  int nnz;
} mmatrix;

int main(int argc, char *argv[]){

  hipsparseStatus_t status;
  hipsparseHandle_t handle=NULL;
  status= hipsparseCreate(&handle);
  hipsolverSpHandle_t handle_cusolver = NULL;
  hipsolverSpCreate(&handle_cusolver);
  hipsparseMatDescr_t descrA; 
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descrA,  HIPSPARSE_INDEX_BASE_ZERO);
  hipblasHandle_t handle_cublas;
  hipblasCreate(&handle_cublas);
  
  int n=1024, *A_i, *A_j;
  double *A_v, *H_rhs, *D_rhs;
  int *H_i, *H_j;
  double *H_v;
  mmatrix *A = (mmatrix *) calloc(1, sizeof(mmatrix));
  mmatrix *H = (mmatrix *) calloc(1, sizeof(mmatrix));
  A->n=n;
  A->m=n;
  A->nnz=2*n-1;
  A->coo_vals =(double*)  calloc(A->nnz, sizeof(double));
  A->csr_ia =(int*)  calloc((A->n)+1, sizeof(int));
  A->coo_cols = (int *)  calloc(A->nnz, sizeof(int));
  H->n=n;
  H->m=n;
  H->nnz=n;
  int totn = (H->n)+(A->n);
  H->coo_vals =(double*)  calloc(H->nnz, sizeof(double));
  H->csr_ia =(int*)  calloc((H->n)+1, sizeof(int));
  H->coo_cols = (int *)  calloc(H->nnz, sizeof(int));
  H_rhs =(double*)  calloc(totn, sizeof(double));
  
  int i;
  //initialize the matrix and the RHS
  A->csr_ia[0]=0;
  for(i=0;i<(A->n);i++){
    if (i){
      A->coo_vals[i*2-1]=i+1;
      A->coo_cols[i*2-1]=i-1;
      A->csr_ia[i]=i*2-1;
    }
    A->coo_vals[i*2]=0;
    A->coo_cols[i*2]=i;
  }
  A->csr_ia[i]=A->nnz;
  for(i=0;i<(H->n);i++){
    H->coo_vals[i]=sqrt(n);
    H->csr_ia[i]=i;
    H->coo_cols[i]=i;
  }
  H->csr_ia[i]=H->nnz;
  for(i=0;i<totn;i++){
    H_rhs[i]=1;
  }

  hipMalloc((void**)&A_v, (A->nnz)*sizeof(double));
  hipMalloc((void**)&A_j, (A->nnz)*sizeof(int));
  hipMalloc((void**)&A_i, ((A->n)+1)*sizeof(int));
  hipMalloc((void**)&H_v, (H->nnz)*sizeof(double));
  hipMalloc((void**)&H_j, (H->nnz)*sizeof(int));
  hipMalloc((void**)&H_i, ((H->n)+1)*sizeof(int));
  hipMalloc((void**)&D_rhs, totn*sizeof(double));

  hipMemcpy(D_rhs, H_rhs, sizeof(double)*totn, hipMemcpyHostToDevice);
  hipMemcpy(A_v, A->coo_vals, sizeof(double)*A->nnz, hipMemcpyHostToDevice);
  hipMemcpy(A_j, A->coo_cols, sizeof(int)*A->nnz, hipMemcpyHostToDevice);
  hipMemcpy(A_i, A->csr_ia, sizeof(int)*((A->n)+1), hipMemcpyHostToDevice);
  hipMemcpy(H_v, H->coo_vals, sizeof(double)*H->nnz, hipMemcpyHostToDevice);
  hipMemcpy(H_j, H->coo_cols, sizeof(int)*H->nnz, hipMemcpyHostToDevice);
  hipMemcpy(H_i, H->csr_ia, sizeof(int)*((H->n)+1), hipMemcpyHostToDevice);

// Test adding to diagonal
  fun_add_diag(A->n, 1.0, A_i, A_j, A_v);

//Transpose A to have its upper triangular part
  double* At_v;
  int *At_i, *At_j;
  hipMalloc(&At_v, (A->nnz)*sizeof(double));
  hipMalloc(&At_j, (A->nnz)*sizeof(int));
  hipMalloc(&At_i, ((A->m)+1)*sizeof(int));
  void *buffercsr=NULL;
  size_t buffersize;
  printf("Transpose A \n");
  status = hipsparseCsr2cscEx2_bufferSize(handle, A->n,A->m, A->nnz,A_v,A_i,A_j,
      At_v,At_i,At_j,HIP_R_64F,HIPSPARSE_ACTION_NUMERIC,HIPSPARSE_INDEX_BASE_ZERO,
      HIPSPARSE_CSR2CSC_ALG1,&buffersize);
  printf("Buffer allocation status %d\n",status);
  hipMalloc(&buffercsr, sizeof(char)*buffersize);
  printf("Buffer size is %d\n",buffersize);
  printf("A dimensions are %d by %d with %d nnz\n", A->n,A->m,A->nnz);
  hipsparseCsr2cscEx2(handle,A->n,A->m, A->nnz,A_v,A_i,A_j,At_v,At_i,At_j,
      HIP_R_64F,
      HIPSPARSE_ACTION_NUMERIC,HIPSPARSE_INDEX_BASE_ZERO,HIPSPARSE_CSR2CSC_ALG1,buffercsr);
  printf("tanspose status %d\n",status);

  hipMemcpy(A->coo_vals, A_v, sizeof(double)*A->nnz, hipMemcpyDeviceToHost);
  hipMemcpy(A->coo_cols, A_j, sizeof(int)*A->nnz, hipMemcpyDeviceToHost);
  hipMemcpy(A->csr_ia, A_i, sizeof(int)*((A->n)+1), hipMemcpyDeviceToHost);
  hipMemcpy(H->coo_vals, H_v, sizeof(double)*H->nnz, hipMemcpyDeviceToHost);
  hipMemcpy(H->coo_cols, H_j, sizeof(int)*H->nnz, hipMemcpyDeviceToHost);
  hipMemcpy(H->csr_ia, H_i, sizeof(int)*((H->n)+1), hipMemcpyDeviceToHost);
  double* Ah_v =(double *) calloc((A->nnz), sizeof(double));
  int* Ah_j =(int *) calloc((A->nnz), sizeof(int));
  int* Ah_i =(int *) calloc((A->m+1), sizeof(int));
  hipMemcpy(Ah_v, At_v, sizeof(double)*A->nnz, hipMemcpyDeviceToHost);
  hipMemcpy(Ah_j, At_j, sizeof(int)*A->nnz, hipMemcpyDeviceToHost);
  hipMemcpy(Ah_i, At_i, sizeof(int)*((A->m)+1), hipMemcpyDeviceToHost);
#if 0
  printf("printing A\n");
  for(i=0; i<n; i++)
  {
    printf("Row %d\n",i);
    for (int j=A->csr_ia[i]; j<A->csr_ia[i+1]; j++)
    {
      printf("Column %d, value %f\n", A->coo_cols[j], A->coo_vals[j]);
    }
  }
#endif
#if 0
  printf("printing H\n");
  for(i=0; i<n; i++)
  {
    printf("Row %d\n",i);
    for (int j=H->csr_ia[i]; j<H->csr_ia[i+1]; j++)
    {
      printf("Column %d, value %f\n", H->coo_cols[j], H->coo_vals[j]);
    }
  }
#endif
#if 0
  printf("printing A transpose\n");
  for(i=0; i<2; i++)
  {
    printf("Row %d\n",i);
    for (int j=Ah_i[i]; j<Ah_i[i+1]; j++)
    {
      printf("Column %d, value %f\n", Ah_j[j], Ah_v[j]);
    }
  }
#endif
double *max_d,*scale;
hipMalloc(&max_d, totn*sizeof(double));
hipMalloc(&scale, totn*sizeof(double));
double* max_h = (double *) calloc(totn, sizeof(double));
for(i=0;i<totn;i++){
  max_h[i]=1; 
}
hipMemcpy(max_d, max_h, sizeof(double)*totn, hipMemcpyHostToDevice);
/*
     This is where the Ruiz magic happens
     */
for(i=0;i<ruiz_its;i++){
  fun_adapt_row_max(n, totn, H_v, H_i, H_j, A_v, A_i, A_j,At_v,At_i, At_j, scale);
  //if(i==ruiz_its-1) flag=0;
  fun_adapt_diag_scale(n, totn, H_v, H_i, H_j, A_v, A_i, A_j, At_v, At_i, At_j,
      scale, D_rhs, &D_rhs[n], max_d);
}
  hipMemcpy(A->coo_vals, A_v, sizeof(double)*A->nnz, hipMemcpyDeviceToHost);
  hipMemcpy(A->coo_cols, A_j, sizeof(int)*A->nnz, hipMemcpyDeviceToHost);
  hipMemcpy(A->csr_ia, A_i, sizeof(int)*((A->n)+1), hipMemcpyDeviceToHost);
  hipMemcpy(H->coo_vals, H_v, sizeof(double)*H->nnz, hipMemcpyDeviceToHost);
  hipMemcpy(H->coo_cols, H_j, sizeof(int)*H->nnz, hipMemcpyDeviceToHost);
  hipMemcpy(H->csr_ia, H_i, sizeof(int)*((H->n)+1), hipMemcpyDeviceToHost);
  hipMemcpy(Ah_v, At_v, sizeof(double)*A->nnz, hipMemcpyDeviceToHost);
  hipMemcpy(Ah_j, At_j, sizeof(int)*A->nnz, hipMemcpyDeviceToHost);
  hipMemcpy(Ah_i, At_i, sizeof(int)*((A->m)+1), hipMemcpyDeviceToHost);
  hipMemcpy(max_h, max_d, sizeof(double)*totn, hipMemcpyDeviceToHost);
  hipMemcpy(H_rhs, D_rhs, sizeof(double)*totn, hipMemcpyDeviceToHost);
#if 0
  printf("max_d\n");
  for(i=0; i<2; i++)
  {
    printf("max_d at row %d = %32.32g\n",i,max_h[i]);
  }
#endif
#if 0
  printf("printing A\n");
  for(i=n-2; i<n; i++)
  {
    printf("Row %d\n",i);
    for (int j=A->csr_ia[i]; j<A->csr_ia[i+1]; j++)
    {
      printf("Column %d, value %f\n", A->coo_cols[j], A->coo_vals[j]);
    }
  }
#endif
#if 0 
  printf("printing H\n");
  for(i=0; i<2; i++)
  {
    printf("Row %d\n",i);
    for (int j=H->csr_ia[i]; j<H->csr_ia[i+1]; j++)
    {
      printf("Column %d, value %f\n", H->coo_cols[j], H->coo_vals[j]);
    }
  }
#endif
#if 0
  printf("printing A transpose\n");
  for(i=n-2; i<n; i++)
  {
    printf("Row %d\n",i);
    for (int j=Ah_i[i]; j<Ah_i[i+1]; j++)
    {
      printf("Column %d, value %f\n", Ah_j[j], Ah_v[j]);
    }
  }
#endif
// Test to compare with MATLAB
  int fails=0;
  if (fabs(H->coo_vals[n/2-1]-0.062378167641326)>tol){
    fails++;
    printf("H not scaled correctly H[n/2-1][n/2-1] = %32.32g\n",H->coo_vals[(H->n)/2-1]);
  }
  if (fabs(A->coo_vals[(A->nnz)-1]-0.005524271728020)>tol){
    fails++;
    printf("A not scaled correctly A[n-1][n-1] = %32.32g\n",A->coo_vals[(A->nnz)-1]);
  }
  if (fabs(Ah_v[1]-0.5)>tol){
    fails++;
    printf("At not scaled correctly At[0][1] = %32.32g \n",Ah_v[1]);
  }
  if (fabs(H_rhs[n/2-1]-0.044151078568835)>tol){
    fails++;
    printf("rhs not scaled correctly H_rhs[n/2-1]= %32.32g\n", H_rhs[n/2-1]);
  }
  if (fabs(max_h[n/2-1]-0.044151078568835)>tol){
    fails++;
    printf("Incorrect scaling factor max_h[n/2-1] = %32.32g\n", max_h[n/2-1]);
  }
  if (fabs(H_rhs[3*n/2-1]-0.044194173824159)>tol){
    fails++;
    printf("rhs not scaled correctly H_rhs[3*n/2-1]= %32.32g\n", H_rhs[3*n/2-1]);
  }
  if (fabs(max_h[3*n/2-1]-0.044194173824159)>tol){
    fails++;
    printf("Incorrect scaling factor max_h[3*n/2-1] = %32.32g\n", max_h[3*n/2-1]);
  }
  if (fails==0) printf("All tests passed\n");
  else 
  {
    printf("%d tests failed\n",fails);
    return 1;
  }
#if 0 //printing
  printf("printing A\n");
  for(i=0; i<n; i++)
  {
    printf("Row %d\n",i);
    for (int j=A->csr_ia[i]; j<A->csr_ia[i+1]; j++)
    {
      printf("Column %d, value %f\n", A->coo_cols[j], A->coo_vals[j]);
    }
  }
#endif
#if 0
  printf("printing A transpose\n");
  for(i=0; i<n; i++)
  {
    printf("Row %d\n",i);
    for (int j=Ah_i[i]; j<Ah_i[i+1]; j++)
    {
      printf("Column %d, value %f\n", Ah_j[j], Ah_v[j]);
    }
  }
#endif
#if 0 //We're no longer concatenating, so no test for this
  //now that matrices are different, test concatenation
  double* C_v;
  int *C_i, *C_j;
  hipMalloc(&C_v, (A->nnz)*2*sizeof(double));
  hipMalloc(&C_j, (A->nnz)*2*sizeof(int));
  hipMalloc(&C_i, ((A->m)*2+1)*sizeof(int));
  blockSize = 32;
  numBlocks = (n + A->m + 1 + blockSize - 1) / blockSize;
  concatenate<<<blockSize,numBlocks>>>(n, n, A->nnz, A->nnz, A_v, A_i, A_j,
   At_v, At_i, At_j, C_v, C_i, C_j);
// Copy back
  double* Ch_v = (double *) calloc((A->nnz)*2, sizeof(double));
  int* Ch_j = (int *) calloc((A->nnz)*2, sizeof(int));
  int* Ch_i = (int *) calloc((A->n)*2+1, sizeof(int));
  hipMemcpy(Ch_v, C_v, sizeof(double)*(A->nnz)*2, hipMemcpyDeviceToHost);
  hipMemcpy(Ch_j, C_j, sizeof(int)*(A->nnz)*2, hipMemcpyDeviceToHost);
  hipMemcpy(Ch_i, C_i, sizeof(int)*((A->n)*2+1), hipMemcpyDeviceToHost);
  for(i=0; i<n*2; i++)
  {
    printf("Row %d\n",i);
    for (int j=Ch_i[i]; j<Ch_i[i+1]; j++)
    {
      printf("Column %d, value %f\n", Ch_j[j], Ch_v[j]);
    }
  }
#endif
  return 0;
}
