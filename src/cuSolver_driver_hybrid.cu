#include "hip/hip_runtime.h"
/* written by SR based on a code by KS
         How to compile:
         nvcc -lcusparse -lcusolver -lcublas cuSolver_driver_chol.cu
 */

#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <assert.h>
#include <ctype.h>
#include <string.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <sys/time.h>
#include <algorithm>
#include "hipsolver.h"
#include <hipsolver.h>
#include <hipsolver.h>
#include <iostream>
#include <memory>
#include <string>
#include "matrix_vector_ops.hpp"
#include "permcheck.hpp"
#include "input_functions.hpp"
#include "schur_complement_cg.hpp"
#include "SchurComplementConjugateGradient.hpp"
#include <RuizClass.hpp>
#define tol 1e-12
#define norm_tol 1e-2
#define ruiz_its 2
// this version reads NORMAL mtx matrices; dont have to be sorted.
int main(int argc, char* argv[])
{
  // Start of block: reading matrices from files and allocating structures for
  // them, to be replaced by HiOp structures
  struct timeval t1, t2;
  double         timeIO = 0.0f, timeM = 0.0f;
  /*** cuda stuff ***/
  hipsparseStatus_t status;
  hipsparseHandle_t handle            = NULL;
  status                             = hipsparseCreate(&handle);
  hipsolverSpHandle_t handle_cusolver = NULL;
  hipsolverSpCreate(&handle_cusolver);
  hipsparseMatDescr_t descrA;
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipblasHandle_t handle_cublas;
  hipblasCreate(&handle_cublas);
  void * dBuffer3 = NULL, *dBuffer4 = NULL;
  size_t bufferSize3 = 0, bufferSize4 = 0;

  // Get matrix block files
  char const* const HFileName  = argv[1];
  char const* const DsFileName = argv[2];
  char const* const JCFileName = argv[3];
  char const* const JDFileName = argv[4];

  // Get rhs block files
  char const* const rxFileName  = argv[5];
  char const* const rsFileName  = argv[6];
  char const* const ryFileName  = argv[7];
  char const* const rydFileName = argv[8];
 // char const* const permFileName = argv[11];
  int skip_lines = atoi(argv[9]);
  double gamma = atof(argv[10]);
  // Matix structure allocations
  // Start block - allocating memory for matrices and vectors
  mmatrix* H  = (mmatrix*)calloc(1, sizeof(mmatrix));
  mmatrix* Ds = (mmatrix*)calloc(1, sizeof(mmatrix));
  mmatrix* JC = (mmatrix*)calloc(1, sizeof(mmatrix));
  mmatrix* JD = (mmatrix*)calloc(1, sizeof(mmatrix));
  // Vector allocations
  double *rx, *rs, *ry, *ryd;

  // read matrices

  read_mm_file_into_coo(HFileName, H, skip_lines);
  sym_coo_to_csr(H);

  read_mm_file_into_coo(DsFileName, Ds, skip_lines);
  coo_to_csr(Ds);

  read_mm_file_into_coo(JCFileName, JC, skip_lines);
  coo_to_csr(JC);

  read_mm_file_into_coo(JDFileName, JD, skip_lines);
  coo_to_csr(JD);
  int jd_flag = (JD->nnz > 0);
  // read right hand side
  rx = (double*)calloc(H->n, sizeof(double));
  read_rhs(rxFileName, rx);
  rs = (double*)calloc(Ds->n, sizeof(double));
  read_rhs(rsFileName, rs);
  ry = (double*)calloc(JC->n, sizeof(double));
  read_rhs(ryFileName, ry);
  ryd = (double*)calloc(JD->n, sizeof(double));
  read_rhs(rydFileName, ryd);
  // now copy data to GPU and format convert
  double *d_rx, *d_rs, *d_ry, *d_ry_c, *d_ryd, *d_ryd_s;
  double *H_a, *Ds_a, *JC_a;
  int *   H_ja, *H_ia;     // columns and rows of H
  int *   JC_ja, *JC_ia;   // columns and rows of JC
  int *   JD_ja, *JD_ia;   // columns and rows of JD
  double *JD_a, *JD_as;
  // allocate space for rhs and copy it to device

  hipMalloc((void**)&d_rx, H->n * sizeof(double));
  hipMalloc((void**)&d_rs, Ds->n * sizeof(double));
  hipMalloc((void**)&d_ry, JC->n * sizeof(double));
  hipMalloc((void**)&d_ry_c, JC->n * sizeof(double));
  hipMalloc((void**)&d_ryd, JD->n * sizeof(double));
  hipMalloc((void**)&d_ryd_s, JD->n * sizeof(double));

  hipMemcpy(d_rx, rx, sizeof(double) * H->n, hipMemcpyHostToDevice);
  hipMemcpy(d_rs, rs, sizeof(double) * Ds->n, hipMemcpyHostToDevice);
  hipMemcpy(d_ry, ry, sizeof(double) * JC->n, hipMemcpyHostToDevice);
  hipMemcpy(d_ry_c, d_ry, sizeof(double) * JC->n, hipMemcpyDeviceToDevice);
  hipMemcpy(d_ryd, ryd, sizeof(double) * JD->n, hipMemcpyHostToDevice);

  // allocate space for matrix and copy it to device
  hipMalloc((void**)&H_a, (H->nnz) * sizeof(double));
  hipMalloc((void**)&H_ja, (H->nnz) * sizeof(int));
  hipMalloc((void**)&H_ia, (H->n + 1) * sizeof(int));

  hipMemcpy(H_a, H->csr_vals, sizeof(double) * H->nnz, hipMemcpyHostToDevice);
  hipMemcpy(H_ia, H->csr_ia, sizeof(int) * (H->n + 1), hipMemcpyHostToDevice);
  hipMemcpy(H_ja, H->csr_ja, sizeof(int) * H->nnz, hipMemcpyHostToDevice);
#if 0 
  printf("CSR H\n");
  for(int i=0; i<10; i++)
  {
    printf("%d\n",i);
    for (int j=H->csr_ia[i]; j<H->csr_ia[i+1]; j++)
    {
      printf("Column %d, value %f\n", H->coo_cols[j], H->coo_vals[j]);
    }
  }
#endif
  hipsparseSpMatDescr_t matH;
  hipsparseCreateCsr(&matH, H->n, H->m, H->nnz, H_ia, H_ja, H_a, HIPSPARSE_INDEX_32I,
    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

  hipMalloc((void**)&Ds_a, (Ds->nnz) * sizeof(double));
  hipMemcpy(Ds_a, Ds->coo_vals, sizeof(double) * Ds->nnz, hipMemcpyHostToDevice);

  hipMalloc((void**)&JC_a, (JC->nnz) * sizeof(double));
  hipMalloc((void**)&JC_ja, (JC->nnz) * sizeof(int));
  hipMalloc((void**)&JC_ia, (JC->n + 1) * sizeof(int));

  hipMemcpy(JC_a, JC->coo_vals, sizeof(double) * JC->nnz, hipMemcpyHostToDevice);
  hipMemcpy(JC_ia, JC->csr_ia, sizeof(int) * (JC->n + 1), hipMemcpyHostToDevice);
  hipMemcpy(JC_ja, JC->coo_cols, sizeof(int) * JC->nnz, hipMemcpyHostToDevice);

  if(jd_flag)
  {
    hipMalloc((void**)&JD_a, (JD->nnz) * sizeof(double));
    hipMalloc((void**)&JD_as, (JD->nnz) * sizeof(double));
    hipMalloc((void**)&JD_ja, (JD->nnz) * sizeof(int));
    hipMalloc((void**)&JD_ia, (JD->n + 1) * sizeof(int));

    hipMemcpy(JD_a, JD->coo_vals, sizeof(double) * JD->nnz, hipMemcpyHostToDevice);
    hipMemcpy(JD_ia, JD->csr_ia, sizeof(int) * (JD->n + 1), hipMemcpyHostToDevice);
    hipMemcpy(JD_ja, JD->coo_cols, sizeof(int) * JD->nnz, hipMemcpyHostToDevice);
  }
  // malloc initial guess (potentially supplied by HiOp)
  // could change at each iteration, but might only happen once

  double* h_x  = (double*)calloc(H->m, sizeof(double));
  double* h_s  = (double*)calloc(Ds->m, sizeof(double));
  double* h_y  = (double*)calloc(JC->n, sizeof(double));
  double* h_yd = (double*)calloc(JD->n, sizeof(double));

  double *d_x, *d_s, *d_y, *d_yd;

  for(int i = 0; i < H->m; i++)
  {
    h_x[i] = 0;
  }

  for(int i = 0; i < Ds->m; i++)
  {
    h_s[i] = 0;
  }

  for(int i = 0; i < JC->n; i++)
  {
    h_y[i] = 0;
  }

  for(int i = 0; i < JD->n; i++)
  {
    h_yd[i] = 0;
  }

  hipMalloc((void**)&d_x, H->m * sizeof(double));
  hipMemcpy(d_x, h_x, sizeof(double) * (H->m), hipMemcpyHostToDevice);
  hipMalloc((void**)&d_s, Ds->m * sizeof(double));
  hipMemcpy(d_s, h_s, sizeof(double) * (Ds->m), hipMemcpyHostToDevice);
  hipMalloc((void**)&d_y, JC->n * sizeof(double));
  hipMemcpy(d_y, h_y, sizeof(double) * (JC->n), hipMemcpyHostToDevice);
  hipMalloc((void**)&d_yd, JD->n * sizeof(double));
  hipMemcpy(d_yd, h_yd, sizeof(double) * (JD->n), hipMemcpyHostToDevice);

  hipsparseSpMatDescr_t matJC;
  hipsparseCreateCsr(&matJC, JC->n, JC->m, JC->nnz, JC_ia, JC_ja, JC_a, HIPSPARSE_INDEX_32I,
    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
  // set up vectors to store products

  double *d_rx_til, *d_rs_til;

  hipMalloc((void**)&d_rx_til, H->n * sizeof(double));
  hipMalloc((void**)&d_rs_til, Ds->n * sizeof(double));
  hipMalloc((void**)&d_ryd_s, JD->n * sizeof(double));
  gettimeofday(&t1, 0);
  hipMemcpy(d_rx_til, d_rx, sizeof(double) * H->n, hipMemcpyDeviceToDevice);
  hipMemcpy(d_rs_til, d_rs, sizeof(double) * Ds->n, hipMemcpyDeviceToDevice);
  gettimeofday(&t2, 0);
  timeM += (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
  hipsparseDnVecDescr_t vec_d_ryd = NULL;
  hipsparseCreateDnVec(&vec_d_ryd, JD->n, d_ryd, HIP_R_64F);
  hipsparseDnVecDescr_t vec_d_rs_til = NULL;
  hipsparseCreateDnVec(&vec_d_rs_til, Ds->n, d_rs_til, HIP_R_64F);

  hipsparseDnVecDescr_t vec_d_ryd_s = NULL;
  hipsparseCreateDnVec(&vec_d_ryd_s, JD->n, d_ryd_s, HIP_R_64F);
  hipsparseDnVecDescr_t vec_d_rx_til = NULL;
  hipsparseCreateDnVec(&vec_d_rx_til, H->n, d_rx_til, HIP_R_64F);
  // Start of block: Setting up eq (4) from the paper
  // start products
  double                one      = 1.0;
  double                zero     = 0.0;
  double                minusone = -1.0;
  int                   nnzHtil;
  double*               Htil_vals = NULL;
  int *                 Htil_cols = NULL, *Htil_rows = NULL;
  hipsparseSpGEMMDescr_t spgemmDesc;
  hipsparseSpGEMM_createDescr(&spgemmDesc);
  hipsparseSpMatDescr_t matJD = NULL;   // create once and overwrite at each iteration
  hipsparseCreateCsr(&matJD, JD->n, JD->m, JD->nnz, JD_ia, JD_ja, JD_a, HIPSPARSE_INDEX_32I,
    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

  hipsparseSpMatDescr_t matJDt = NULL;
  double* JDt_a;
  int *   JDt_ja, *JDt_ia;
  if(jd_flag)   // if JD is not all zeros (otherwise computation is saved)
  {
    // Creating a CSR matrix and buffer for transposing - done only once
    hipMalloc(&JDt_a, (JD->nnz) * sizeof(double));
    hipMalloc(&JDt_ja, (JD->nnz) * sizeof(int));
    hipMalloc(&JDt_ia, ((JD->m) + 1) * sizeof(int));
    void*  buffercsr = NULL;
    size_t buffersize;
    hipsparseCsr2cscEx2_bufferSize(handle, JD->n, JD->m, JD->nnz, JD_a, JD_ia, JD_ja, JDt_a, JDt_ia,
      JDt_ja, HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1,
      &buffersize);
    hipMalloc(&buffercsr, sizeof(char) * buffersize);
    // Applying the transpose to the matrix - done every iteration
    gettimeofday(&t1, 0);
    hipsparseCsr2cscEx2(handle, JD->n, JD->m, JD->nnz, JD_a, JD_ia, JD_ja, JDt_a, JDt_ia, JDt_ja,
      HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1,
      buffercsr);
    gettimeofday(&t2, 0);
    timeM += (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
    hipsparseCreateCsr(&matJDt, JD->m, JD->n, JD->nnz, JDt_ia, JDt_ja, JDt_a, HIPSPARSE_INDEX_32I,
      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    // math ops for eq (4) done at every iteration
    gettimeofday(&t1, 0);
    fun_row_scale(JD->n, JD_a, JD_ia, JD_ja, JD_as, d_ryd, d_ryd_s, Ds_a);
    gettimeofday(&t2, 0);
    timeM += (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
    hipsparseSpMatDescr_t matJDs = NULL;   //(except this part)
    hipsparseCreateCsr(&matJDs, JD->n, JD->m, JD->nnz, JD_ia, JD_ja, JD_as, HIPSPARSE_INDEX_32I,
      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    gettimeofday(&t1, 0);
    fun_add_vecs(JD->n, d_ryd_s, one, d_rs);
    // create buffer for matvec - done once
    /*
      size_t bufferSize_rx = 0;
      hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_TRANSPOSE, &one, matJD, vec_d_ryd_s, &one,
        vec_d_rx_til, HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize_rx);
      void* buffer_rx = NULL;
      hipMalloc(&buffer_rx, bufferSize_rx);
      printf("bufferSize_rx is %d", bufferSize_rx); //this is 0
      */
    // matvec done every iteration
    fun_SpMV(one, matJDt, vec_d_ryd_s, one, vec_d_rx_til);
    gettimeofday(&t2, 0);
    timeM += (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
    // Compute H_til= H+J_d^T * D_s * J_d
    // Allocating for SPGEMM - done once
    hipsparseSpMatDescr_t matJDtDxJD = NULL;
    hipsparseCreateCsr(&matJDtDxJD, JD->m, JD->m, 0, NULL, NULL, NULL, HIPSPARSE_INDEX_32I,
      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    // ask bufferSize3 bytes for external memory
    hipsparseSpGEMM_workEstimation(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
      HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matJDt, matJDs, &zero, matJDtDxJD, HIP_R_64F,
      HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &bufferSize3, NULL);
    hipMalloc((void**)&dBuffer3, bufferSize3);
    // inspect the matrices A and B to understand the memory requirement for
    // the next step
    hipsparseSpGEMM_workEstimation(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
      HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matJDt, matJDs, &zero, matJDtDxJD, HIP_R_64F,
      HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &bufferSize3, dBuffer3);
    hipsparseSpGEMM_compute(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
      HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matJDt, matJDs, &zero, matJDtDxJD, HIP_R_64F,
      HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &bufferSize4, NULL);
    hipMalloc((void**)&dBuffer4, bufferSize4);
    // SPGEMM - done every iteration
    gettimeofday(&t1, 0);
    hipsparseSpGEMM_compute(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
      HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matJDt, matJDs, &zero, matJDtDxJD, HIP_R_64F,
      HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &bufferSize4, dBuffer4);
    gettimeofday(&t2, 0);
    timeM += (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;

    // compute the intermediate product of A * B - happens once
    int64_t JDtDxJD_num_rows1, JDtDxJD_num_cols1, JDtDxJD_nnz1;
    hipsparseSpMatGetSize(matJDtDxJD, &JDtDxJD_num_rows1, &JDtDxJD_num_cols1, &JDtDxJD_nnz1);
    int *   JDtDxJD_rows, *JDtDxJD_cols;
    double* JDtDxJD_vals;
    hipMalloc((void**)&JDtDxJD_rows, (JDtDxJD_num_rows1 + 1) * sizeof(int));
    hipMalloc((void**)&JDtDxJD_cols, JDtDxJD_nnz1 * sizeof(int));
    hipMalloc((void**)&JDtDxJD_vals, JDtDxJD_nnz1 * sizeof(double));
    // SPGEMM - happens every iteration
    hipsparseCsrSetPointers(matJDtDxJD, JDtDxJD_rows, JDtDxJD_cols, JDtDxJD_vals);
  gettimeofday(&t1, 0);
    hipsparseSpGEMM_copy(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
      &one, matJDt, matJDs, &zero, matJDtDxJD, HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc);
  gettimeofday(&t2, 0);
  timeM += (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
    /* It's time for the sum Htilde= H + (J_d^TD_xJ_d)
     nnzTotalDevHostPtr points to host memory
     Allocation for matrix addition - happens once
    */
    size_t bufferSizeInBytes_add;
    void*  buffer_add         = NULL;
    int*   nnzTotalDevHostPtr = &nnzHtil;
    hipMalloc((void**)&Htil_rows, sizeof(int) * ((H->n) + 1));
    hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
    hipsparseDcsrgeam2_bufferSizeExt(handle, H->n, H->n, &one, descrA, H->nnz, H_a, H_ia, H_ja, &one,
      descrA, JDtDxJD_nnz1, JDtDxJD_vals, JDtDxJD_rows, JDtDxJD_cols, descrA, Htil_vals, Htil_rows,
      Htil_cols, &bufferSizeInBytes_add);
    hipMalloc((void**)&buffer_add, sizeof(char) * bufferSizeInBytes_add);
    hipsparseXcsrgeam2Nnz(handle, H->n, H->n, descrA, H->nnz, H_ia, H_ja, descrA, JDtDxJD_nnz1,
      JDtDxJD_rows, JDtDxJD_cols, descrA, Htil_rows, nnzTotalDevHostPtr, buffer_add);
    nnzHtil = *nnzTotalDevHostPtr;
    hipMalloc((void**)&Htil_cols, sizeof(int) * (nnzHtil));
    hipMalloc((void**)&Htil_vals, sizeof(double) * (nnzHtil));
    // Matrix addition, happens every iteration
    gettimeofday(&t1, 0);
    hipsparseDcsrgeam2(handle, H->n, H->n, &one, descrA, H->nnz, H_a, H_ia, H_ja, &one, descrA,
      JDtDxJD_nnz1, JDtDxJD_vals, JDtDxJD_rows, JDtDxJD_cols, descrA, Htil_vals, Htil_rows,
      Htil_cols, buffer_add);
    gettimeofday(&t2, 0);
    timeM += (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
    // Free only happens at last iteration
    hipFree(JDtDxJD_rows);
    hipFree(JDtDxJD_cols);
    hipFree(JDtDxJD_vals);
    hipFree(buffercsr);
    hipFree(buffer_add);
  }   // This closes the if J_d!=0 statement
  else
  {   // overwite H with Htil if JD==0
    hipMalloc((void**)&Htil_rows, sizeof(int) * ((H->n) + 1));
    hipMalloc((void**)&Htil_cols, sizeof(int) * (H->nnz));
    hipMalloc((void**)&Htil_vals, sizeof(double) * (H->nnz));
    gettimeofday(&t1, 0);
    hipMemcpy(Htil_vals, H_a, sizeof(double) * (H->nnz), hipMemcpyDeviceToDevice);
    gettimeofday(&t2, 0);
    timeM += (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
    hipMemcpy(Htil_rows, H_ia, sizeof(int) * (H->n + 1), hipMemcpyDeviceToDevice);
    hipMemcpy(Htil_cols, H_ja, sizeof(int) * (H->nnz), hipMemcpyDeviceToDevice);
    nnzHtil = H->nnz;
  }
  // Start of block: Ruiz scaling
  // Allocation - happens once
  int     nHJ = (H->n) + (JC->n);
  double* JCt_a;
  int *   JCt_ja, *JCt_ia;
  hipMalloc(&JCt_a, (JC->nnz) * sizeof(double));
  hipMalloc(&JCt_ja, (JC->nnz) * sizeof(int));
  hipMalloc(&JCt_ia, ((JC->m) + 1) * sizeof(int));
  void*  buffercsr3 = NULL;
  size_t buffersize3;
  hipsparseCsr2cscEx2_bufferSize(handle, JC->n, JC->m, JC->nnz, JC_a, JC_ia, JC_ja, JCt_a, JCt_ia,
    JCt_ja, HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1,
    &buffersize3);
  hipMalloc(&buffercsr3, sizeof(char) * buffersize3);
  // Transpose JC - happens every iteration
    gettimeofday(&t1, 0);
  hipsparseCsr2cscEx2(handle, JC->n, JC->m, JC->nnz, JC_a, JC_ia, JC_ja, JCt_a, JCt_ia, JCt_ja,
    HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1,
    buffercsr3);
    gettimeofday(&t2, 0);
    timeM += (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
  hipsparseSpMatDescr_t matJCt = NULL;
  hipsparseCreateCsr(&matJCt, JC->m, JC->n, JC->nnz, JCt_ia, JCt_ja, JCt_a, HIPSPARSE_INDEX_32I,
    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
#if 1 //this block is only activated to check solution (requires more copying)
  // saves the original JC and JCt  
  double*               JC_a_c = NULL;
  int *                 JC_ia_c = NULL, *JC_ja_c = NULL;
  hipMalloc((void**)&JC_ia_c, sizeof(int) * ((JC->n) + 1));
  hipMalloc((void**)&JC_ja_c, sizeof(int) * (JC->nnz));
  hipMalloc((void**)&JC_a_c, sizeof(double) * (JC->nnz));
  hipMemcpy(JC_a_c, JC_a, sizeof(double) * (JC->nnz), hipMemcpyDeviceToDevice);
  hipMemcpy(JC_ia_c, JC_ia, sizeof(int) * (JC->n + 1), hipMemcpyDeviceToDevice);
  hipMemcpy(JC_ja_c, JC_ja, sizeof(int) * (JC->nnz), hipMemcpyDeviceToDevice);
  hipsparseSpMatDescr_t matJC_c = NULL;
  hipsparseCreateCsr(&matJC_c, JC->n, JC->m, JC->nnz, JC_ia_c, JC_ja_c, JC_a_c, HIPSPARSE_INDEX_32I,
    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
  double*               JCt_a_c = NULL;
  int *                 JCt_ia_c = NULL, *JCt_ja_c = NULL;
  hipMalloc((void**)&JCt_ia_c, sizeof(int) * ((JC->m) + 1));
  hipMalloc((void**)&JCt_ja_c, sizeof(int) * (JC->nnz));
  hipMalloc((void**)&JCt_a_c, sizeof(double) * (JC->nnz));
  hipMemcpy(JCt_a_c, JCt_a, sizeof(double) * (JC->nnz), hipMemcpyDeviceToDevice);
  hipMemcpy(JCt_ia_c, JCt_ia, sizeof(int) * ((JC->m) + 1), hipMemcpyDeviceToDevice);
  hipMemcpy(JCt_ja_c, JCt_ja, sizeof(int) * (JC->nnz), hipMemcpyDeviceToDevice);
  hipsparseSpMatDescr_t matJCt_c = NULL;
  hipsparseCreateCsr(&matJCt_c, JC->m, JC->n, JC->nnz, JCt_ia_c, JCt_ja_c, JCt_a_c, HIPSPARSE_INDEX_32I,
    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
#endif
#if 0 
  double *JC_a_h;
  int *JC_ia_h, *JC_ja_h;
  JC_a_h = (double*)calloc(JC->nnz, sizeof(double));
  JC_ja_h = (int*)calloc(JC->nnz, sizeof(int));
  JC_ia_h = (int*)calloc((JC->n)+1, sizeof(int));
  hipMemcpy(JC_a_h, JC_a_c, sizeof(double)*(JC->nnz), hipMemcpyDeviceToHost);
  hipMemcpy(JC_ja_h, JC_ja_c, sizeof(int)*(JC->nnz), hipMemcpyDeviceToHost);
  hipMemcpy(JC_ia_h, JC_ia_c, sizeof(int)*((JC->n)+1), hipMemcpyDeviceToHost);
  printf("CSR J_c\n");
  for(int i=(JC->n)-2; i<(JC->n); i++)
  {
    printf("%d\n",i);
    for (int j=JC_ia_h[i]; j<JC_ia_h[i+1]; j++)
    {
      printf("Column %d, value %f\n", JC_ja_h[j], JC_a_h[j]);
    }
  }
  free(JC_a_h);
  free(JC_ia_h);
  free(JC_ja_h);
#endif
  // setup vectors for scaling
#if 0 //class implementation
RuizClass hjr(H->n, nHJ, Htil_vals, Htil_rows, Htil_cols, JC_a,
    JC_ia, JC_ja, JCt_a, JCt_ia, JCt_ja, d_rx_til, d_ry);
hjr.setup();
hjr.init_max_d();
for(int i=0;i<ruiz_its;i++){
  hjr.row_max();
  hjr.diag_scale();
}
double* max_d;
max_d = hjr.get_max_d();
#endif
#if 1 //function implemention
  // Allocation - happens once
  double *max_d, *scale;
  hipMalloc(&max_d, nHJ * sizeof(double));
  hipMalloc(&scale, nHJ * sizeof(double));
  double* max_h = (double*)calloc(nHJ, sizeof(double));
  // Initialization and actual scaling - happen every iteration
  for(int i = 0; i < nHJ; i++)
  {
    max_h[i] = 1;
  }
  gettimeofday(&t1, 0);
  hipMemcpy(max_d, max_h, sizeof(double) * nHJ, hipMemcpyHostToDevice);
  for(int i = 0; i < ruiz_its; i++)
  {
    fun_adapt_row_max(H->n, nHJ, Htil_vals, Htil_rows, Htil_cols, JC_a, JC_ia,
        JC_ja, JCt_a, JCt_ia, JCt_ja, scale);
    fun_adapt_diag_scale(H->n, nHJ, Htil_vals, Htil_rows, Htil_cols, JC_a,
      JC_ia, JC_ja, JCt_a, JCt_ia, JCt_ja, scale, d_rx_til, d_ry, max_d);
  }
  gettimeofday(&t2, 0);
  timeM += (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
#endif
#if 0 
  double *Ht_a_h;
  int *Ht_ia_h, *Ht_ja_h;
  Ht_a_h = (double*)calloc(nnzHtil, sizeof(double));
  Ht_ja_h = (int*)calloc(nnzHtil, sizeof(int));
  Ht_ia_h = (int*)calloc((H->n)+1, sizeof(int));
  hipMemcpy(Ht_a_h, Htil_vals, sizeof(double)*(nnzHtil), hipMemcpyDeviceToHost);
  hipMemcpy(Ht_ja_h, Htil_cols, sizeof(int)*(nnzHtil), hipMemcpyDeviceToHost);
  hipMemcpy(Ht_ia_h, Htil_rows, sizeof(int)*((H->n)+1), hipMemcpyDeviceToHost);
  printf("CSR H\n");
  for(int i=(H->n)-2; i<(H->n); i++)
  {
    printf("%d\n",i);
    for (int j=Ht_ia_h[i]; j<Ht_ia_h[i+1]; j++)
    {
      printf("Column %d, value %f\n", Ht_ja_h[j], Ht_a_h[j]);
    }
  }
  free(Ht_a_h);
  free(Ht_ia_h);
  free(Ht_ja_h);
#endif
#if 0
  hipMemcpy(max_h,max_d, sizeof(double)*nHJ, hipMemcpyDeviceToHost);
  for (int i=0;i<10;i++)
  printf("D[%d] = %f\n", i, max_h[i]);
#endif
  // Start of block, setting up eq (5)
  // Allocation for SPGEMM - happens once
  hipsparseSpMatDescr_t matJCtJC = NULL;
  hipsparseCreateCsr(&matJCtJC, JC->m, JC->m, 0, NULL, NULL, NULL, HIPSPARSE_INDEX_32I,
    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
  void*  bufferJC = NULL;
  size_t buffersizeJC;
  // ask bufferSize1 bytes for external memory
  hipsparseSpGEMM_workEstimation(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
    HIPSPARSE_OPERATION_NON_TRANSPOSE, &gamma, matJCt, matJC, &zero, matJCtJC, HIP_R_64F,
    HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &buffersizeJC, NULL);
  hipMalloc((void**)&bufferJC, buffersizeJC);
  // inspect the matrices A and B to understand the memory requirement for
  // the next step
  hipsparseSpGEMM_workEstimation(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
    HIPSPARSE_OPERATION_NON_TRANSPOSE, &gamma, matJCt, matJC, &zero, matJCtJC, HIP_R_64F,
    HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &buffersizeJC, bufferJC);
  void*  bufferJC2 = NULL;
  size_t buffersizeJC2;
  hipsparseSpGEMM_compute(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
    &gamma, matJCt, matJC, &zero, matJCtJC, HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc,
    &buffersizeJC2, NULL);
  hipMalloc((void**)&bufferJC2, buffersizeJC2);
  // compute the intermediate product of A * B
  // Compute SPGEMM - done every iteration
  gettimeofday(&t1, 0);
  hipsparseSpGEMM_compute(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
    HIPSPARSE_OPERATION_NON_TRANSPOSE, &gamma, matJCt, matJC, &zero, matJCtJC, HIP_R_64F,
    HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &buffersizeJC2, bufferJC2);
  gettimeofday(&t2, 0);
  timeM += (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
  // Allocation - happens once
  int64_t JCtJC_num_rows1, JCtJC_num_cols1, JCtJC_nnz1;
  hipsparseSpMatGetSize(matJCtJC, &JCtJC_num_rows1, &JCtJC_num_cols1, &JCtJC_nnz1);
  int *   JCtJC_rows, *JCtJC_cols;
  double* JCtJC_vals;
  hipMalloc((void**)&JCtJC_rows, (JC->m + 1) * sizeof(int));
  hipMalloc((void**)&JCtJC_cols, JCtJC_nnz1 * sizeof(int));
  hipMalloc((void**)&JCtJC_vals, JCtJC_nnz1 * sizeof(double));
  // SPGEMM - happens very iterations
  hipsparseCsrSetPointers(matJCtJC, JCtJC_rows, JCtJC_cols, JCtJC_vals);
  gettimeofday(&t1, 0);
  hipsparseSpGEMM_copy(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
      &gamma, matJCt, matJC, &zero, matJCtJC, HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc);
  gettimeofday(&t2, 0);
  timeM += (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
#if 0
  int *JCtJC_i, *JCtJC_j;
  double *JCtJC_v;
  JCtJC_i=(int*) malloc((H->n + 1)*sizeof(int));
  JCtJC_j=(int*) malloc((JCtJC_nnz1)*sizeof(int));
  JCtJC_v=(double*) malloc((JCtJC_nnz1)*sizeof(double));
    
  hipMemcpy(JCtJC_v, JCtJC_vals, sizeof(double)*JCtJC_nnz1, hipMemcpyDeviceToHost);
  hipMemcpy(JCtJC_i, JCtJC_rows, sizeof(int)*(H->n+1), hipMemcpyDeviceToHost);  
  hipMemcpy(JCtJC_j, JCtJC_cols, sizeof(int)*JCtJC_nnz1, hipMemcpyDeviceToHost);
  printf("gamma*J_c^TJ_c num rows = %d, nnz = %d\n",H->n, JCtJC_nnz1);
  for(int i=3000; i<3001; i++)
  {
    printf("Row %d starts at place %d\n",i,JCtJC_i[i]);
    for (int j=JCtJC_i[i]; j<JCtJC_i[i+1]; j++)
    {
      printf("Column %d value %f\n", JCtJC_j[j], JCtJC_v[j]);
    }
  }
  free(JCtJC_i);
  free(JCtJC_j);
  free(JCtJC_v);
#endif
  /* It's time for the sum Hgamma= Htilde + gamma(J_c^TJ_c)
   nnzTotalDevHostPtr2 points to host memory
   Allocation for matrix addition - happens once*/
  size_t  bufferSizeInBytes_add2;
  char*   buffer_add2 = NULL;
  int     nnzHgam;
  int*    nnzTotalDevHostPtr2 = &nnzHgam;
  double* Hgam_vals           = NULL;
  int *   Hgam_cols = NULL, *Hgam_rows = NULL;
  hipMalloc((void**)&Hgam_rows, sizeof(int) * ((H->n) + 1));
  hipsparseDcsrgeam2_bufferSizeExt(handle, H->n, H->n, &one, descrA, nnzHtil, Htil_vals, Htil_rows,
    Htil_cols, &one, descrA, JCtJC_nnz1, JCtJC_vals, JCtJC_rows, JCtJC_cols, descrA, Hgam_vals,
    Hgam_rows, Hgam_cols, &bufferSizeInBytes_add2);
  hipMalloc((void**)&buffer_add2, sizeof(char) * bufferSizeInBytes_add2);
  hipsparseXcsrgeam2Nnz(handle, H->n, H->n, descrA, nnzHtil, Htil_rows, Htil_cols, descrA,
    JCtJC_nnz1, JCtJC_rows, JCtJC_cols, descrA, Hgam_rows, nnzTotalDevHostPtr2, buffer_add2);
  nnzHgam = *nnzTotalDevHostPtr2;
  printf("nnzHgam = %d\n", nnzHgam);
  hipMalloc((void**)&Hgam_cols, sizeof(int) * (nnzHgam));
  hipMalloc((void**)&Hgam_vals, sizeof(double) * (nnzHgam));
  // Matrix addition - happens every iteration
  gettimeofday(&t1, 0);
  hipsparseDcsrgeam2(handle, H->n, H->n, &one, descrA, nnzHtil, Htil_vals, Htil_rows, Htil_cols,
    &one, descrA, JCtJC_nnz1, JCtJC_vals, JCtJC_rows, JCtJC_cols, descrA, Hgam_vals, Hgam_rows,
    Hgam_cols, buffer_add2);
  gettimeofday(&t2, 0);
  timeM += (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
  double* d_rx_hat;
  hipMalloc((void**)&d_rx_hat, H->n * sizeof(double));
  gettimeofday(&t1, 0);
  hipMemcpy(d_rx_hat, d_rx_til, sizeof(double) * H->n, hipMemcpyDeviceToDevice);
  gettimeofday(&t2, 0);
  timeM += (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
  hipsparseDnVecDescr_t vec_d_rx_hat = NULL;
  hipsparseCreateDnVec(&vec_d_rx_hat, H->n, d_rx_hat, HIP_R_64F);
  hipsparseDnVecDescr_t vec_d_ry = NULL;
  hipsparseCreateDnVec(&vec_d_ry, JC->n, d_ry, HIP_R_64F);
  /* this size is 0 anyways
  size_t  bufferSize_rx_hat = 0;
  hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &gamma, matJCt, vec_d_ry, &one,
    vec_d_rx_hat, HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize_rx_hat);
  void* buffer_rx_hat = NULL;
  hipMalloc(&buffer_rx_hat, bufferSize_rx_hat);
  */
  gettimeofday(&t1, 0);
  fun_SpMV(gamma, matJCt, vec_d_ry, one, vec_d_rx_hat);
  gettimeofday(&t2, 0);
  timeM += (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
  // Start of block: permutation calculation (happens once)
  int *Hgam_h_rows, *Hgam_h_cols;
  Hgam_h_rows = (int*)malloc((H->n + 1) * sizeof(int));
  Hgam_h_cols = (int*)malloc((nnzHgam) * sizeof(int));
  hipMemcpy(Hgam_h_rows, Hgam_rows, sizeof(int) * (H->n + 1), hipMemcpyDeviceToHost);
  hipMemcpy(Hgam_h_cols, Hgam_cols, sizeof(int) * nnzHgam, hipMemcpyDeviceToHost);
#if 0
  double *Hgam_h_vals;
  Hgam_h_vals=(double*) malloc((nnzHgam)*sizeof(double));
    
  hipMemcpy(Hgam_h_vals, Hgam_vals, sizeof(double)*nnzHgam, hipMemcpyDeviceToHost);
  printf("Hgam num rows = %d, nnz = %d\n",H->n, nnzHgam);
  for(int i=500; i<502; i++)
  {
    printf("Row %d\n",i);
    for (int j=Hgam_h_rows[i]; j<Hgam_h_rows[i+1]; j++)
    {
      printf("Column %d, value %f\n", Hgam_h_cols[j], Hgam_h_vals[j]);
    }
  }
  free(Hgam_h_vals);
#endif
  int* perm       = NULL;
  int* rev_perm   = NULL;
  int* perm_mapH  = NULL;
  int* perm_mapJ  = NULL;
  int* perm_mapJt = NULL;
  perm            = (int*)calloc(H->n, sizeof(int));
  perm_mapH       = (int*)calloc(nnzHgam, sizeof(int));
  perm_mapJ       = (int*)calloc(JC->nnz, sizeof(int));
  perm_mapJt      = (int*)calloc(JC->nnz, sizeof(int));
  rev_perm        = (int*)calloc(H->n, sizeof(int));
  cusolverSpXcsrsymamdHost(handle_cusolver, H->n, nnzHgam, descrA, Hgam_h_rows, Hgam_h_cols,
    perm);   // overwriting perm in next line for test
#if 0
  printf("Overwriting permutation \n");
  int *MLperm=(int*)  calloc(H->n, sizeof(int));
  read_1idx_perm(permFileName, MLperm);
  perm=MLperm;
#endif
  int *Hgam_p_rows, *Hgam_p_cols;
  Hgam_p_rows = (int*)malloc((H->n + 1) * sizeof(int));
  Hgam_p_cols = (int*)malloc((nnzHgam) * sizeof(int));
  reverse_perm(H->n, perm, rev_perm);
  make_vec_map_rc(
    H->n, Hgam_h_rows, Hgam_h_cols, perm, rev_perm, Hgam_p_rows, Hgam_p_cols, perm_mapH);

  int* Jc_p_cols;
  Jc_p_cols = (int*)malloc((JC->nnz) * sizeof(int));
  make_vec_map_c(JC->n, JC->csr_ia, JC->coo_cols, rev_perm, Jc_p_cols, perm_mapJ);

  int* Jct_p_cols;
  int* Jct_p_rows;
  Jct_p_cols = (int*)malloc((JC->nnz) * sizeof(int));
  Jct_p_rows = (int*)malloc((JC->m + 1) * sizeof(int));
  int* Jct_cols;
  int* Jct_rows;
  Jct_cols = (int*)malloc((JC->nnz) * sizeof(int));
  Jct_rows = (int*)malloc((JC->m + 1) * sizeof(int));
  hipMemcpy(Jct_rows, JCt_ia, sizeof(int) * (JC->m + 1), hipMemcpyDeviceToHost);
  hipMemcpy(Jct_cols, JCt_ja, sizeof(int) * (JC->nnz), hipMemcpyDeviceToHost);
  make_vec_map_r(JC->m, Jct_rows, Jct_cols, perm, Jct_p_rows, Jct_p_cols, perm_mapJt);

  hipMemcpy(Hgam_rows, Hgam_p_rows, sizeof(int) * (H->n + 1), hipMemcpyHostToDevice);
  hipMemcpy(Hgam_cols, Hgam_p_cols, sizeof(int) * nnzHgam, hipMemcpyHostToDevice);
  hipMemcpy(JCt_ja, Jct_p_cols, sizeof(int) * (JC->nnz), hipMemcpyHostToDevice);
  hipMemcpy(JCt_ia, Jct_p_rows, sizeof(int) * (JC->m + 1), hipMemcpyHostToDevice);
  hipMemcpy(JC_ja, Jc_p_cols, sizeof(int) * (JC->nnz), hipMemcpyHostToDevice);

  int *drev_perm, *d_perm, *d_perm_mapH, *d_perm_mapJ, *d_perm_mapJt;
  hipMalloc(&drev_perm, (H->n) * sizeof(int));
  hipMalloc(&d_perm, (H->n) * sizeof(int));
  hipMalloc(&d_perm_mapH, (nnzHgam) * sizeof(int));
  hipMalloc(&d_perm_mapJ, (JC->nnz) * sizeof(int));
  hipMalloc(&d_perm_mapJt, (JC->nnz) * sizeof(int));
  hipMemcpy(drev_perm, rev_perm, sizeof(int) * (H->n), hipMemcpyHostToDevice);
  hipMemcpy(d_perm, perm, sizeof(int) * (H->n), hipMemcpyHostToDevice);
  hipMemcpy(d_perm_mapH, perm_mapH, sizeof(int) * nnzHgam, hipMemcpyHostToDevice);
  hipMemcpy(d_perm_mapJ, perm_mapJ, sizeof(int) * (JC->nnz), hipMemcpyHostToDevice);
  hipMemcpy(d_perm_mapJt, perm_mapJt, sizeof(int) * (JC->nnz), hipMemcpyHostToDevice);

  double *Hgamp_val, *Jcp_val, *Jctp_val;
  hipMalloc(&Hgamp_val, (nnzHgam) * sizeof(double));
  hipMalloc(&Jcp_val, (JC->nnz) * sizeof(double));
  hipMalloc(&Jctp_val, (JC->nnz) * sizeof(double));
  // Start of block: permutation application - happens every iteration
  gettimeofday(&t1, 0);
  fun_map_idx(nnzHgam, d_perm_mapH, Hgam_vals, Hgamp_val);
  fun_map_idx(JC->nnz, d_perm_mapJ, JC_a, Jcp_val);
  fun_map_idx(JC->nnz, d_perm_mapJt, JCt_a, Jctp_val);
  gettimeofday(&t2, 0);
  timeM += (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
  printf("time for forming Hgamma ev(ms). : %16.16f\n", timeM);
  hipsparseSpMatDescr_t matJCp = NULL;
  hipsparseCreateCsr(&matJCp, JC->n, JC->m, JC->nnz, JC_ia, JC_ja, Jcp_val, HIPSPARSE_INDEX_32I,
    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
#if 1
  gettimeofday(&t1, 0);
  fun_add_diag(H->n, zero, Hgam_rows, Hgam_cols, Hgamp_val);
  gettimeofday(&t2, 0);
  timeIO = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
  printf("time for forming Hdelta ev(ms). : %16.16f\n", timeIO);
#endif
#if 0
  hipMemcpy(JC->coo_vals, Jcp_val, sizeof(double)*(JC->nnz), hipMemcpyDeviceToHost);
  hipMemcpy(JC->csr_ia, JC_ia, sizeof(int)*(JC->n+1), hipMemcpyDeviceToHost);  
  hipMemcpy(JC->coo_cols, JC_ja, sizeof(int)*(JC->nnz), hipMemcpyDeviceToHost);
  printf("JC num rows = %d, nnz = %d\n",JC->n, JC->nnz);
  for(int i=1099; i<1100; i++)
  {
    printf("Row %d\n",i);
    for (int j=JC->csr_ia[i]; j<JC->csr_ia[i+1]; j++)
    {
      printf("Column %d, value %f\n", JC->coo_cols[j], JC->coo_vals[j]);
    }
  }
#endif
#if 0
  double *Hgamp_h_vals;
  Hgamp_h_vals=(double*) malloc((nnzHgam)*sizeof(double));
  hipMemcpy(Hgamp_h_vals, Hgamp_val, sizeof(double)*nnzHgam, hipMemcpyDeviceToHost);
  printf("Hgamp num rows = %d, nnz = %d\n",H->n, nnzHgam);
  for(int i=1099; i<1100; i++)
  {
    printf("Row %d starts at place %d\n",i,Hgam_p_rows[i]);
    for (int j=Hgam_p_rows[i]; j<Hgam_p_rows[i+1]; j++)
    {
      printf("Column %d, value %f\n", Hgam_p_cols[j], Hgamp_h_vals[j]);
    }
  }
  free(Hgamp_h_vals);
#endif
  hipsparseSpMatDescr_t matJCtp = NULL;
  hipsparseCreateCsr(&matJCtp, JC->m, JC->n, JC->nnz, JCt_ia, JCt_ja, Jctp_val, HIPSPARSE_INDEX_32I,
    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
#if 0
  double *JCth_v;
  int *JCth_i, *JCth_j;
  JCth_v=(double*) malloc((JC->nnz)*sizeof(double));
  JCth_j=(int*) malloc((JC->nnz)*sizeof(int));
  JCth_i=(int*) malloc((JC->m+1)*sizeof(int));
  hipMemcpy(JCth_v, Jctp_val, sizeof(double)*(JC->nnz), hipMemcpyDeviceToHost);
  hipMemcpy(JCth_j, JCt_ja, sizeof(int)*(JC->nnz), hipMemcpyDeviceToHost);
  hipMemcpy(JCth_i, JCt_ia, sizeof(int)*(JC->m +1), hipMemcpyDeviceToHost);
  printf("JCt num rows = %d, nnz = %d\n",JC->m, JC->nnz);
  for(int i=1099; i<1100; i++)
  {
    printf("Row %d starts at place %d\n",i,JCth_i[i]);
    for (int j=JCth_i[i]; j<JCth_i[i+1]; j++)
    {
      printf("Column %d, value %f\n", JCth_j[j], JCth_v[j]);
    }
  }
  free(JCth_v);
  free(JCth_i);
  free(JCth_j);
#endif

  double* d_rxp;
  hipMalloc((void**)&d_rxp, H->n * sizeof(double));
  fun_map_idx(H->n, d_perm, d_rx_hat, d_rxp);
  //  Start of block: Factorization of Hgamma
  //  Symbolic analysis: Happens once
  csrcholInfo_t info = NULL;
  cusolverSpCreateCsrcholInfo(&info);
  gettimeofday(&t1, 0);
  cusolverSpXcsrcholAnalysis(handle_cusolver, H->n, nnzHgam, descrA, Hgam_rows, Hgam_cols, info);
  size_t internalDataInBytes, workspaceInBytes;
  cusolverSpDcsrcholBufferInfo(handle_cusolver, H->n, nnzHgam, descrA, Hgamp_val, Hgam_rows,
    Hgam_cols, info, &internalDataInBytes, &workspaceInBytes);
  gettimeofday(&t2, 0);
  timeIO = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
  void* buffer_gpu = NULL;
  hipMalloc(&buffer_gpu, sizeof(char) * workspaceInBytes);
  printf("time for symbolic analysis ev(ms). : %16.16f\n", timeIO);
  int singularity = 0;
  gettimeofday(&t1, 0);
  // Numerical factorization - happens every iteration
  cusolverSpDcsrcholFactor(
    handle_cusolver, H->n, nnzHgam, descrA, Hgamp_val, Hgam_rows, Hgam_cols, info, buffer_gpu);
  gettimeofday(&t2, 0);
  timeIO = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
  cusolverSpDcsrcholZeroPivot(handle_cusolver, info, tol, &singularity);
  printf("time for factorization analysis ev(ms). : %16.16f\n", timeIO);
  if(singularity >= 0)
  {
    fprintf(stderr, "Error: H is not invertible, singularity=%d\n", singularity);
    return 1;
  }
  else
    printf("matrix nonsingular, proceed\n");
  //  Start of block : setting up the right hand side for equation 7
  //  Allocation - happens once
  double* d_Hrxp;
  hipMalloc((void**)&d_Hrxp, H->n * sizeof(double));
  double* d_schur;
  hipMalloc((void**)&d_schur, JC->n * sizeof(double));
  //  Solve and copy - happen every iteration
  cusolverSpDcsrcholSolve(handle_cusolver, H->n, d_rxp, d_Hrxp, info, buffer_gpu);
  hipMemcpy(d_schur, d_ry, sizeof(double) * JC->n, hipMemcpyDeviceToDevice);
#if 0
  printf("printing ry\n");
  hipMemcpy(ry, d_schur, sizeof(double)*(JC->n), hipMemcpyDeviceToHost);
  for (int i=(JC->n)-10; i<JC->n; i++){
    printf("ry[%d] = %f\n", i, ry[i]);
  }
#endif
  // Allocation - happens once
  hipsparseDnVecDescr_t vec_d_schur = NULL;
  hipsparseCreateDnVec(&vec_d_schur, JC->n, d_schur, HIP_R_64F);
  hipsparseDnVecDescr_t vec_d_Hrxp = NULL;
  hipsparseCreateDnVec(&vec_d_Hrxp, H->n, d_Hrxp, HIP_R_64F);
  /* this is just zero
  size_t               bufferSize_schur = 0;
  hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matJCp, vec_d_Hrxp,
    &minusone, vec_d_schur, HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize_schur);
  void* buffer_schur = NULL;
  hipMalloc(&buffer_schur, bufferSize_schur);
  */
  //  Matrix vector multiply - happens every iteration
  fun_SpMV(one, matJCp, vec_d_Hrxp, minusone, vec_d_schur);
#if 0 
  double *h_schur;
  h_schur=(double*) malloc((JC->n)*sizeof(double));
  hipMemcpy(h_schur, d_schur, sizeof(double)*(JC->n), hipMemcpyDeviceToHost);
  for (int i=15; i<485; i++){
     printf("schur[%d] = %f\n", i, h_schur[i]);
  }
  free(h_schur);
#endif
  // Start of block - conjugate gradient on eq (7)
  // Solving eq (7) via CG - happens every iteration
  //function implementation
#if 0
  int itmax = (JC->n) / 10;
  schur_cg(matJCp, matJCtp, info, d_y, d_schur, itmax, tol, JC->n, JC->m, JC->nnz,
    buffer_gpu, handle, handle_cusolver, handle_cublas);
#endif
  // class implementation
#if 1
  SchurComplementConjugateGradient sccg(
      matJCp, matJCtp, info, d_y, d_schur, JC->n, JC->m, JC->nnz,buffer_gpu);
  sccg.allocate();
  sccg.setup();
  sccg.solve();
#endif
#if 0
  hipMemcpy(h_y, d_y, sizeof(double)*(JC->n), hipMemcpyDeviceToHost);
  for (int i=(JC->n)-10; i<JC->n; i++){
     printf("y[%d] = %f\n", i, h_y[i]);
  }
#endif
  // Start of block - recovering the solution to the original system by parts
  // this part is to recover delta_x
  // Allocation - happens once
  hipsparseDnVecDescr_t vec_d_y = NULL;
  hipsparseCreateDnVec(&vec_d_y, JC->n, d_y, HIP_R_64F);
  hipsparseDnVecDescr_t vec_d_rxp = NULL;
  hipsparseCreateDnVec(&vec_d_rxp, H->n, d_rxp, HIP_R_64F);
  /* this is zero anyways
  size_t               bufferSize_d_z = 0;
  hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minusone, matJCtp, vec_d_y,
    &one, vec_d_rxp, HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize_d_z);
  void* buffer_d_z = NULL;
  hipMalloc(&buffer_d_z, bufferSize_d_z);
  */
  // Matrix-vector product - happens every iteration
  gettimeofday(&t1, 0);
  fun_SpMV(minusone, matJCtp, vec_d_y, one, vec_d_rxp);
  //  Allocation - happens once
  double* d_z;
  hipMalloc((void**)&d_z, H->n * sizeof(double));
  //  Solve - happens every iteration
  cusolverSpDcsrcholSolve(handle_cusolver, H->n, d_rxp, d_z, info, buffer_gpu);
  fun_map_idx(H->n, drev_perm, d_z, d_x);
#if 0
  double *h_rx_hat;
  printf("delta_x\n");
  h_rx_hat=(double*) malloc((H->n)*sizeof(double));
  hipMemcpy(h_rx_hat, d_x, sizeof(double)*(H->n), hipMemcpyDeviceToHost);
  for (int i=(H->n)-10; i<H->n; i++){
    printf("delta_x[%d] = %f\n", i, h_rx_hat[i]);
  }
  free(h_rx_hat);
#endif
  // scale back delta_y and delta_x (every iteration)
  fun_vec_scale(H->n, d_x, max_d);
  fun_vec_scale(JC->n, d_y, &max_d[H->n]);
#if 0 
  hipMemcpy(h_x, d_x, sizeof(double)*(H->n), hipMemcpyDeviceToHost);
  for (int i=(H->n)-10; i<H->n; i++){
     printf("x[%d] = %f\n", i, h_x[i]);
  }
#endif

#if 0   // check max_d
  hipMemcpy(max_h, max_d, sizeof(double)*(nHJ), hipMemcpyDeviceToHost);
  for (int i=0; i<10; i++){
     printf("max[%d] = %f\n", i, max_h[i]);
  }
  for (int i=H->n; i<H->n+10; i++){
     printf("max[%d] = %f\n", i, max_h[i]);
  }
#endif
  // now recover delta_s and delta_yd
  //  Allocation - happens once
  hipsparseDnVecDescr_t vec_d_x = NULL;
  hipsparseCreateDnVec(&vec_d_x, H->n, d_x, HIP_R_64F);
  hipsparseDnVecDescr_t vec_d_s = NULL;
  hipsparseCreateDnVec(&vec_d_s, Ds->n, d_s, HIP_R_64F);
  hipMemcpy(d_s, d_ryd, sizeof(double) * (Ds->m), hipMemcpyDeviceToDevice);
  if(jd_flag)
  {
    /* This is zero anyways
    size_t               bufferSize_dx = 0;
    hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matJD, vec_d_x,
      &minusone, vec_d_ryd, HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize_dx);
    void* buffer_dx = NULL;
    hipMalloc(&buffer_dx, bufferSize_dx);
    hipFree(buffer_dx);
    */
    //  Matrix-vector product - happens every iteration
    fun_SpMV(one, matJD, vec_d_x, minusone, vec_d_s);
  }
  else
  {   //  Math operations - happens every iteration
    fun_mult_const(Ds->n, minusone, d_s);
  }
  //  Math operations - happens every iteration
  hipMemcpy(d_yd, d_s, sizeof(double) * (Ds->m), hipMemcpyDeviceToDevice);
  fun_vec_scale(Ds->n, d_yd, Ds_a);
  fun_add_vecs(Ds->n, d_yd, minusone, d_rs);
  gettimeofday(&t2, 0);
  timeIO = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
  printf("time for recovering solution ev(ms). : %16.16f\n", timeIO);
#if 0 
  hipMemcpy(h_yd, d_yd, sizeof(double)*(JD->n), hipMemcpyDeviceToHost);
  for (int i=(JD->n)-10; i<JD->n; i++){
     printf("yd[%d] = %f\n", i, h_yd[i]);
  }
  hipMemcpy(h_s, d_s, sizeof(double)*(Ds->n), hipMemcpyDeviceToHost);
  for (int i=(Ds->n)-10; i<Ds->n; i++){
     printf("s[%d] = %f\n", i, h_s[i]);
  }
#endif
  //  Start of block, calculate error of Ax-b 
  //  Calculate error in rx
  gettimeofday(&t1, 0);
  double norm_rx_sq=0, norm_rs_sq=0, norm_ry_sq=0, norm_ryd_sq=0;
  double norm_resx_sq=0, norm_resy_sq=0; 
  // This will aggregate the squared norms of the residual and rhs
  // Note that by construction the residuals of rs and ryd are 0
  hipblasDdot(handle_cublas, H->n, d_rx, 1, d_rx, 1, &norm_rx_sq);
  hipblasDdot(handle_cublas, Ds->n, d_rs, 1, d_rs, 1, &norm_rs_sq);
  hipblasDdot(handle_cublas, JC->n, d_ry_c, 1, d_ry_c, 1, &norm_ry_sq);
  hipblasDdot(handle_cublas, JD->n, d_ryd, 1, d_ryd, 1, &norm_ryd_sq);
  norm_rx_sq+= norm_rs_sq + norm_ry_sq + norm_ryd_sq;
  hipsparseDnVecDescr_t vec_d_rx = NULL;
  hipsparseCreateDnVec(&vec_d_rx, H->n, d_rx, HIP_R_64F);
  hipsparseDnVecDescr_t vec_d_yd = NULL;
  hipsparseCreateDnVec(&vec_d_yd, JD->n, d_yd, HIP_R_64F);
  fun_SpMV(minusone, matH, vec_d_x, one, vec_d_rx);
  hipblasDdot(handle_cublas, H->n, d_rx, 1, d_rx, 1, &norm_resx_sq);
  if (jd_flag){
    fun_SpMV(minusone, matJDt, vec_d_yd, one, vec_d_rx);
    hipblasDdot(handle_cublas, H->n, d_rx, 1, d_rx, 1, &norm_resx_sq);
  }
#if 0 
  double *JCt_a_h;
  int *JCt_ia_h, *JCt_ja_h;
  JCt_a_h = (double*)calloc(JC->nnz, sizeof(double));
  JCt_ja_h = (int*)calloc(JC->nnz, sizeof(int));
  JCt_ia_h = (int*)calloc((JC->m)+1, sizeof(int));
  hipMemcpy(JCt_a_h, JCt_a_c, sizeof(double)*(JC->nnz), hipMemcpyDeviceToHost);
  hipMemcpy(JCt_ja_h, JCt_ja_c, sizeof(int)*(JC->nnz), hipMemcpyDeviceToHost);
  hipMemcpy(JCt_ia_h, JCt_ia_c, sizeof(int)*((JC->m)+1), hipMemcpyDeviceToHost);
  printf("CSR J_c\n");
  for(int i=1500; i<1502; i++)
  {
    printf("%d\n",i);
    for (int j=JCt_ia_h[i]; j<JCt_ia_h[i+1]; j++)
    {
     printf("Column %d, value %f\n", JCt_ja_h[j], JCt_a_h[j]);
    }
  }
  free(JCt_a_h);
  free(JCt_ia_h);
  free(JCt_ja_h);
#endif
#if 0
  hipMemcpy(h_y, d_y, sizeof(double)*(JC->n), hipMemcpyDeviceToHost);
  for (int i=(JC->n)-10; i<JC->n; i++){
     printf("y[%d] = %f\n", i, h_y[i]);
  }
#endif
  fun_SpMV(minusone, matJCt_c, vec_d_y, one, vec_d_rx);
  hipblasDdot(handle_cublas, H->n, d_rx, 1, d_rx, 1, &norm_resx_sq);
  //  Calculate error in ry
  hipsparseDnVecDescr_t vec_d_ry_c = NULL;
  hipsparseCreateDnVec(&vec_d_ry_c, JC->n, d_ry_c, HIP_R_64F);
  fun_SpMV(minusone, matJC_c, vec_d_x, one, vec_d_ry_c);
  hipblasDdot(handle_cublas, JC->n, d_ry_c, 1, d_ry_c, 1, &norm_resy_sq);
  // Calculate final relative norm
  norm_resx_sq+=norm_resy_sq;
  double norm_res = sqrt(norm_resx_sq)/sqrt(norm_rx_sq);
  printf("||Ax-b||/||b|| = %32.32g\n", norm_res);
  //  Start of block - free memory
  free(rx);
  free(rs);
  free(ry);
  free(ryd);
  hipFree(d_x);
  hipFree(d_s);
  hipFree(d_y);
  hipFree(d_yd);
  hipFree(d_z);
  hipFree(d_rx);
  hipFree(d_rxp);
  hipFree(d_Hrxp);
  hipFree(d_schur);
  hipFree(d_rs);
  hipFree(d_ry);
  hipFree(d_ry_c);
  hipFree(d_ryd);
  hipFree(d_ryd_s);
  hipFree(d_rx_til);
  hipFree(d_rx_hat);
  hipFree(d_rs_til);
  hipFree(H_a);
  hipFree(H_ja);
  hipFree(H_ia);
  hipFree(Ds_a);
  hipFree(JC_a);
  hipFree(JC_ja);
  hipFree(JC_ia);
  hipFree(JCt_a);
  hipFree(JCt_ja);
  hipFree(JCt_ia);
  hipFree(JC_a_c);
  hipFree(JC_ja_c);
  hipFree(JC_ia_c);
  hipFree(JCt_a_c);
  hipFree(JCt_ja_c);
  hipFree(JCt_ia_c);
  hipFree(JD_a);
  hipFree(JD_as);
  hipFree(JD_ja);
  hipFree(JD_ia);
  hipFree(JDt_a);
  hipFree(JDt_ja);
  hipFree(JDt_ia);
  free(h_x);
  free(h_s);
  free(h_y);
  free(h_yd);
  free(H->csr_ia);
  free(H->csr_ja);
  free(H->csr_vals);
  free(H->coo_cols);
  free(H->coo_rows);
  free(H->coo_vals);
  free(H);
  free(Ds->coo_cols);
  free(Ds->csr_ia);
  free(Ds->coo_rows);
  free(Ds->coo_vals);
  free(Ds);
  free(JC->coo_cols);
  free(JC->csr_ia);
  free(JC->coo_rows);
  free(JC->coo_vals);
  free(JC);
  free(JD->coo_cols);
  free(JD->csr_ia);
  free(JD->coo_rows);
  free(JD->coo_vals);
  free(JD);
  hipFree(buffercsr3);
  hipFree(buffer_gpu);
  // hipFree(buffer_schur);
  // hipFree(buffer_d_z);
  hipFree(dBuffer3);
  hipFree(dBuffer4);
  hipFree(buffer_add2);
  // hipFree(buffer_rx_hat);
  hipFree(bufferJC);
  hipFree(bufferJC2);
  hipFree(max_d);
#if 0
  free(max_h);
  hipFree(scale);
#endif
  hipFree(d_perm);
  hipFree(drev_perm);
  hipFree(d_perm_mapH);
  hipFree(d_perm_mapJ);
  hipFree(d_perm_mapJt);
  hipFree(Htil_rows);
  hipFree(Htil_cols);
  hipFree(Htil_vals);
  hipFree(JCtJC_rows);
  hipFree(JCtJC_cols);
  hipFree(JCtJC_vals);
  hipFree(Hgam_rows);
  hipFree(Hgam_cols);
  hipFree(Hgam_vals);
  hipFree(Hgamp_val);
  hipFree(Jcp_val);
  hipFree(Jctp_val);
  free(Hgam_h_rows);
  free(Hgam_h_cols);
  free(Hgam_p_rows);
  free(Hgam_p_cols);
  free(Jc_p_cols);
  free(Jct_p_cols);
  free(Jct_rows);
  free(Jct_cols);
  free(Jct_p_rows);
  gettimeofday(&t2, 0);
  timeIO = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
  printf("time for IO+API+error ev(ms). : %16.16f\n", timeIO);
  if (norm_res<norm_tol){
    printf("Residual test passed ");
  }
  else{
    printf("Residual test failed ");
    return 1;
  }
  return 0;
}
