#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <sys/time.h>
#include <algorithm>
#include "hipsolver.h"
#include <hipsolver.h>
#include <hipsolver.h>

#include "SchurComplementConjugateGradient.hpp"
  // parametrized constructor
  SchurComplementConjugateGradient::SchurComplementConjugateGradient(
      hipsparseSpMatDescr_t matJC, hipsparseSpMatDescr_t matJCt, csrcholInfo_t dH,
      double* x0, double* b, int n, int m, int nnz, void* buffer_gpu) :
      matJC_(matJC), matJCt_(matJCt), dH_(dH), x0_(x0), b_(b), n_(n), m_(m),
      nnz_(nnz), buffer_gpu_(buffer_gpu){}

  // destructor
  SchurComplementConjugateGradient::~SchurComplementConjugateGradient(){
  free(ycp);
  hipFree(y);
  hipFree(z);
  hipFree(r);
  hipFree(w);
  hipFree(p);
  hipFree(s);
  };

  // solver API
  void SchurComplementConjugateGradient::allocate(){
  hipsparseCreate(&handle);  
  hipsolverSpCreate(&handle_cusolver);
  hipblasCreate(&handle_cublas);
  ycp = (double*)calloc(m_, sizeof(double));
  for(int i = 0; i < m_; i++)
  {
    ycp[i] = 0;
  }
  hipMalloc((void**)&y, m_ * sizeof(double));
  hipMalloc((void**)&z, m_ * sizeof(double));
  hipMalloc((void**)&r, n_ * sizeof(double));
  hipMalloc((void**)&w, n_ * sizeof(double));
  hipMalloc((void**)&p, n_ * sizeof(double));
  hipMalloc((void**)&s, n_ * sizeof(double));

  //  Allocation - happens once
  hipsparseCreateDnVec(&vecx, n_, x0_, HIP_R_64F);
  hipsparseCreateDnVec(&vecb, n_, b_, HIP_R_64F);
  hipsparseCreateDnVec(&vecy, m_, y, HIP_R_64F);
  hipsparseCreateDnVec(&vecz, m_, z, HIP_R_64F);
  hipsparseCreateDnVec(&vecr, n_, r, HIP_R_64F);
  hipsparseCreateDnVec(&vecw, n_, w, HIP_R_64F);
  hipsparseCreateDnVec(&vecp, n_, p, HIP_R_64F);
  hipsparseCreateDnVec(&vecs, n_, s, HIP_R_64F);

  }
  void SchurComplementConjugateGradient::setup(){
  hipMemcpy(y, ycp, sizeof(double) * (m_), hipMemcpyHostToDevice);
  hipMemcpy(z, y, sizeof(double) * (m_), hipMemcpyDeviceToDevice);
  hipMemcpy(r, b_, sizeof(double) * (n_), hipMemcpyDeviceToDevice);
  hipMemcpy(w, b_, sizeof(double) * (n_), hipMemcpyDeviceToDevice);
  hipMemcpy(p, r, sizeof(double) * (n_), hipMemcpyDeviceToDevice);
  hipMemcpy(s, w, sizeof(double) * (n_), hipMemcpyDeviceToDevice);
  
  }
  int SchurComplementConjugateGradient::solve(){
  gettimeofday(&t1, 0);
  fun_SpMV(one, matJCt_, vecx, zero, vecy);
  cusolverSpDcsrcholSolve(handle_cusolver, m_, y, z, dH_, buffer_gpu_);
  fun_SpMV(minusone, matJC_, vecz, one, vecr);
  double gam_i;
  hipblasDdot(handle_cublas, n_, r, 1, r, 1, &gam_i);
  fun_SpMV(one, matJCt_, vecr, zero, vecy);
  cusolverSpDcsrcholSolve(handle_cusolver, m_, y, z, dH_, buffer_gpu_);
  fun_SpMV(one, matJC_, vecz, zero, vecw);
  double beta = 0, delta, alpha, gam_i1;
  hipblasDdot(handle_cublas, n_, w, 1, r, 1, &delta);
  alpha           = gam_i / delta;
  double minalpha = -alpha;
  int i;
  for(i = 0; i < itmax_; i++)
  {
    hipblasDscal(handle_cublas, n_, &beta, p, 1);
    hipblasDaxpy(handle_cublas, n_, &one, r, 1, p, 1);

    hipblasDscal(handle_cublas, n_, &beta, s, 1);
    hipblasDaxpy(handle_cublas, n_, &one, w, 1, s, 1);

    hipblasDaxpy(handle_cublas, n_, &alpha, p, 1, x0_, 1);
    minalpha = -alpha;
    hipblasDaxpy(handle_cublas, n_, &minalpha, s, 1, r, 1);

    hipblasDdot(handle_cublas, n_, r, 1, r, 1, &gam_i1);
    if(sqrt(gam_i1) < tol_)
    {
      gettimeofday(&t2, 0);
      timeIO = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
      printf("time for CG ev(ms). : %16.16f\n", timeIO);
      printf("Convergence occured at iteration %d\n", i);
      break;
    }
    // product with w=Ar starts here
    fun_SpMV(one, matJCt_, vecr, zero, vecy);
    cusolverSpDcsrcholSolve(handle_cusolver, m_, y, z, dH_, buffer_gpu_);
    fun_SpMV(one, matJC_, vecz, zero, vecw);

    hipblasDdot(handle_cublas, n_, w, 1, r, 1, &delta);
    beta  = gam_i1 / gam_i;
    gam_i = gam_i1;
    alpha = gam_i / (delta - beta * gam_i / alpha);
  }
  printf("Error is %32.32g \n", sqrt(gam_i1));
  if (i==itmax_){
    gettimeofday(&t2, 0);
    timeIO = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;
    printf("time for CG ev(ms). : %16.16f\n", timeIO);
    printf("No CG convergence in %d iterations\n", itmax_);
    return 1;
  }
    return 0;
  }
  void SchurComplementConjugateGradient::set_solver_tolerance(double tol)
  {
    tol_ = tol;
  }
  void SchurComplementConjugateGradient::set_solver_itmax(int itmax)
  {
    itmax_ = itmax;
  }

