#include <stdio.h>
#include <stdlib.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <sys/time.h>
#include <algorithm>
#include "hipsolver.h"
#include <hipsolver.h>
#include <hipsolver.h>

#include "RuizScale.hpp"
  // parametrized constructor
  RuizClass::RuizClass(int n, int m, double* H_v, int* H_i, int* H_j,
      double* J_v, int* J_i, int* J_j, double* Jt_v, int* Jt_i, int* Jt_j,
      double* rhs_1, double* rhs_2) :
      n_(n), m_(m), H_v_(H_v), H_i_(H_i), H_j_(H_j), 
      J_v_(J_v), J_i_(J_i), J_j_(J_j), Jt_v_(J_v), Jt_i_(J_i), Jt_j_(J_j),
      rhs_1_(rhs_1), rhs_2_(rhs_2){}

  // destructor
  RuizScale::~RuizScale(){
  hipFree(scale);
  };

  //  Initialization
  void RuizScale::row_max(){

  }
  void RuizScale::row_max(){

  }
